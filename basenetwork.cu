#include "hip/hip_runtime.h"
/*
 * BaseNetwork
 * ネットワークの親クラス
 * 
 * 2017.11.20
 * mutsuki
 */
#include "basenetwork.h"
#include "matrix_function.h"
#include "hyperparameter.h"
#include "cuda_common.h"
#include <iostream>

using namespace mtk;
template<class T>
__global__ void deviceMap(float *device_ptr_dst,float* device_ptr_src,float a,int max_t){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(max_t <= tid)
		return;
	device_ptr_dst[tid] = T(a)(device_ptr_src[tid]);
}

class AdagradMake{
	float s;
public:
	__device__ AdagradMake(float s):s(s){}
	__device__ float operator()(float x){
		return 1.0f/(sqrtf(x)+s);
	}
};
class MaxAndInverse{
	float m;
public:
	__device__ MaxAndInverse(float m):m(m){}
	__device__ float operator() (float l) const{
		return 1.0f/fmaxf(fabsf(m),fabsf(l));
	}
};

BaseNetwork::BaseNetwork(int input_size,int output_size,int batch_size,std::string network_name,hipblasHandle_t cublas,float learning_rate,float adagrad_epsilon,float attenuation_rate):
	input_size(input_size),output_size(output_size),batch_size(batch_size),network_name(network_name),cublas(cublas),learning_rate(learning_rate),adagrad_epsilon(adagrad_epsilon),attenuation_rate(attenuation_rate)
{
	w1.setSize(output_size,input_size)->allocateDevice()->initDeviceRandom(-1.0f,1.0f);
	dw1.setSize(output_size,input_size)->allocateDevice()->initDeviceConstant(0.0f);
	rdw1.setSize(output_size,input_size)->allocateDevice()->initDeviceConstant(0.0f);
	b1.setSize(output_size,1)->allocateDevice()->initDeviceRandom(-1.0f,1.0f);
	db1.setSize(output_size,1)->allocateDevice()->initDeviceConstant(0.0f);
	rdb1.setSize(output_size,1)->allocateDevice()->initDeviceConstant(0.0f);
	u1.setSize(output_size,batch_size)->allocateDevice()->initDeviceConstant(0.0f);
	z0.setSize(input_size,batch_size)->allocateDevice()->initDeviceConstant(0.0f);
	adagrad_w1.setSize(output_size,input_size)->allocateDevice()->initDeviceConstant(0.0f);
	adagrad_b1.setSize(output_size,1)->allocateDevice()->initDeviceConstant(0.0f);
	all1_b.setSize(1,batch_size)->allocateDevice()->initDeviceConstant(1.0f);
	all1_o.setSize(1,output_size)->allocateDevice()->initDeviceConstant(1.0f);
	all1_i.setSize(1,input_size)->allocateDevice()->initDeviceConstant(1.0f);
	u.setSize(output_size,batch_size)->allocateDevice()->initDeviceConstant(0.0f);
	max_b_i.setSize(output_size,1)->allocateDevice()->initDeviceConstant(0.0f);
	max_w_i.setSize(output_size,input_size)->allocateDevice()->initDeviceConstant(0.0f);
	b1_tmp.setSize(output_size,1)->allocateDevice()->initDeviceConstant(0.0f);
	w1_tmp.setSize(output_size,input_size)->allocateDevice()->initDeviceConstant(0.0f);
	std::cout<<network_name<<"("<<input_size<<","<<output_size<<","<<batch_size<<")"<<std::endl;
	std::cout<<" - learning rate = "<<learning_rate<<std::endl;
	std::cout<<" - adagrad epsilon = "<<adagrad_epsilon<<std::endl;
	std::cout<<" - momentum rate = "<<attenuation_rate<<std::endl;
}

BaseNetwork::~BaseNetwork(){}

void BaseNetwork::learningForwardPropagation(mtk::MatrixXf &output,const mtk::MatrixXf& input){
	const float one = 1.0f,zero = 0.0f;
	mtk::MatrixFunction::copy(cublas, z0, input);
	CUBLAS_HANDLE_ERROR(hipblasSgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N,
			output_size,batch_size,1,
			&one,
			b1.getDevicePointer(),b1.getRows(),
			all1_b.getDevicePointer(),1,
			&zero,
			u1.getDevicePointer(),u1.getRows()));
	CUBLAS_HANDLE_ERROR(hipblasSgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N,
			output_size,batch_size,input_size,
			&one,
			w1.getDevicePointer(),w1.getRows(),
			input.getDevicePointer(),input.getRows(),
			&one,
			u1.getDevicePointer(),output_size));
	this->learningActivation(output,u1);
}


void BaseNetwork::learningReflect(){
	const float one = 1.0f;
	const float minus_learning_rate = -learning_rate;
	mtk::MatrixFunction::elementwiseProduct(cublas,adagrad_w1,rdw1,rdw1,1.0f,1.0f);
	mtk::MatrixFunction::elementwiseProduct(cublas,adagrad_b1,rdb1,rdb1,1.0f,1.0f);
	// dw1を作る
	mtk::MatrixFunction::map<AdagradMake>(w1_tmp,adagrad_w1,adagrad_epsilon);
	mtk::MatrixFunction::elementwiseProduct(cublas,dw1,rdw1,w1_tmp,minus_learning_rate,attenuation_rate);
	// db1を作る
	mtk::MatrixFunction::map<AdagradMake>(b1_tmp,adagrad_b1,adagrad_epsilon);
	mtk::MatrixFunction::elementwiseProduct(cublas,db1,rdb1,b1_tmp,minus_learning_rate,attenuation_rate);

	// 更新
	CUBLAS_HANDLE_ERROR( hipblasSaxpy( cublas, w1.getSize(),
				&one,
				dw1.getDevicePointer(),1,
				w1.getDevicePointer(),1) );
	CUBLAS_HANDLE_ERROR( hipblasSaxpy( cublas, b1.getSize(),
				&one,
				db1.getDevicePointer(),1,
				b1.getDevicePointer(),1) );
	
	// 重みが大きくなりすぎないように
	int max_w_index = 0;
	float zero = 0.0f;
	// 絶対値が最大の要素のindexを返す
	CUBLAS_HANDLE_ERROR( hipblasIsamax( cublas,w1.getSize(),
				w1.getDevicePointer(),1,&max_w_index) );
	CUBLAS_HANDLE_ERROR( hipblasSgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N,
				1,output_size,1,
				&one,
				w1.getDevicePointer()+max_w_index,1,
				all1_o.getDevicePointer(),1,
				&zero,
				max_b_i.getDevicePointer(),1));
	mtk::MatrixFunction::map<MaxAndInverse>(max_b_i,max_b_i,1.0f);
	CUBLAS_HANDLE_ERROR( hipblasSgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N,
				output_size,input_size,1,
				&one,
				max_b_i.getDevicePointer(),output_size,
				all1_i.getDevicePointer(),1,
				&zero,
				max_w_i.getDevicePointer(),max_w_i.getRows()));
	// 正規化
	mtk::MatrixFunction::elementwiseProduct(cublas,w1_tmp,max_w_i,w1);
	mtk::MatrixFunction::elementwiseProduct(cublas,b1_tmp,max_b_i,b1);
	// 結果をコピー
	mtk::MatrixFunction::copy(cublas,w1,w1_tmp);
	mtk::MatrixFunction::copy(cublas,b1,b1_tmp);
}

void BaseNetwork::learningBackPropagation(mtk::MatrixXf &next_error, const mtk::MatrixXf &d2){
	mtk::MatrixFunction::copy(cublas,next_error,d2);
	mtk::MatrixFunction::copy(cublas,d1,d2);
}

mtk::MatrixXf* BaseNetwork::getWeightPointer(){return &w1;}
mtk::MatrixXf* BaseNetwork::getBiasPointer(){return &b1;}
int BaseNetwork::getInputSize(){return input_size;}
int BaseNetwork::getOutputSize(){return output_size;}
std::string BaseNetwork::getNetworkName(){return network_name;}


// test methods

void BaseNetwork::testForwardPropagation(mtk::MatrixXf &output,const mtk::MatrixXf &input) {
	const float one = 1.0f,zero = 0.0f;
	CUBLAS_HANDLE_ERROR(hipblasSgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N,
			output_size,test_batch_size,1,
			&one,
			b1.getDevicePointer(),b1.getRows(),
			all1_t.getDevicePointer(),1,
			&zero,
			test_u.getDevicePointer(),test_u.getRows()));
	CUBLAS_HANDLE_ERROR(hipblasSgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N,
			output_size,test_batch_size,input_size,
			&one,
			w1.getDevicePointer(),w1.getRows(),
			input.getDevicePointer(),input.getRows(),
			&one,
			test_u.getDevicePointer(),test_u.getRows()));

	this->testActivation(output,test_u);
}

void BaseNetwork::testInit(int b){
	test_batch_size = b;
	test_u.setSize(output_size,test_batch_size)->allocateDevice()->initDeviceConstant(0.0f);
	all1_t.setSize(1,test_batch_size)->allocateDevice()->initDeviceConstant(1.0f);
}

void BaseNetwork::testRelease(){
	test_u.releaseDevice();
	all1_t.releaseDevice();
}

