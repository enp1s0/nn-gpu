
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void cudaHandleError( hipError_t err,const char *file,int line ) {
	if (err != hipSuccess) {
		printf( "CUDA Error\n%s in %s at line %d\n", hipGetErrorString( err ),file, line );
		exit( EXIT_FAILURE );
	}
}
__host__ __device__ inline int threads_ceildiv(int size,int blocks){
	return (blocks + size - 1)/blocks;
}
