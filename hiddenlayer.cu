#include "hip/hip_runtime.h"
#include "hiddenlayer.h"
#include "cuda_common.h"
#include "activation.h"
#include "matrix_function.h"
#include <iostream>

using namespace mtk;

const int BLOCKS = 1 << 7;

template<class T>
__global__ void deviceMap(float *device_ptr_dst,float* device_ptr_src,int max_t){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(max_t <= tid)
		return;
	device_ptr_dst[tid] = T()(device_ptr_src[tid]);
}
__global__ void devicePointwiseProduct(float *device_ptr_dst,float* device_ptr_src0,float* device_ptr_src1,int max_t){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(max_t <= tid)
		return;
	device_ptr_dst[tid] = device_ptr_src0[tid] * device_ptr_src1[tid];
}




HiddenLayer::HiddenLayer(int input_size,int output_size,int batch_size,std::string layer_name,hipblasHandle_t cublas,float learning_rate,float adagrad_epsilon,float annuation_rate):
	BaseLayer(input_size,output_size,batch_size,layer_name,cublas,learning_rate,adagrad_epsilon,annuation_rate)
{
}

HiddenLayer::~HiddenLayer(){
}
//HiddenLayer::~HiddenLayer(){}

void HiddenLayer::learningBackPropagation(mtk::MatrixXf &next_error, const mtk::MatrixXf &d2, const mtk::MatrixXf *w2){
	int u1_size = u1.getRows() * u1.getCols();
	const float one = 1.0f,zero = 0.0f;
	deviceMap<dActReLU><<<BLOCKS,threads_ceildiv(u1.getSize(),BLOCKS)>>>(u1.getDevicePointer(),u1.getDevicePointer(),u1.getSize());
	CUBLAS_HANDLE_ERROR(hipblasSgemm(cublas,HIPBLAS_OP_T,HIPBLAS_OP_N,
			output_size,batch_size,w2->getRows(),
			&one,
			w2->getDevicePointer(),w2->getRows(),
			d2.getDevicePointer(),d2.getRows(),
			&zero,
			u.getDevicePointer(),u.getRows()));
	mtk::MatrixFunction::elementwiseProduct(cublas,next_error,u,u1);
	float alpha = 1.0f/batch_size;
	CUBLAS_HANDLE_ERROR(hipblasSgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_T,
			output_size,input_size,batch_size,
			&alpha,
			next_error.getDevicePointer(),next_error.getRows(),
			z0.getDevicePointer(),z0.getRows(),
			&zero,
			rdw1.getDevicePointer(),rdw1.getRows()));
	CUBLAS_HANDLE_ERROR(hipblasSgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N,
			output_size,1,batch_size,
			&alpha,
			next_error.getDevicePointer(),next_error.getRows(),
			all1_b.getDevicePointer(),z0.getRows(),
			&zero,
			rdb1.getDevicePointer(),rdb1.getRows()));
}

void HiddenLayer::activation(mtk::MatrixXf &output, const mtk::MatrixXf &input) const {
	deviceMap<dActReLU><<<BLOCKS,threads_ceildiv(input.getSize(),BLOCKS)>>>(output.getDevicePointer(),input.getDevicePointer(),input.getSize());
}
