#include "hiddennetwork.h"
#include "cuda_common.h"
#include "activation.h"
#include "matrix_function.h"
#include <iostream>

using namespace mtk;


HiddenNetwork::HiddenNetwork(int input_size,int output_size,int batch_size,std::string network_name,hipblasHandle_t cublas,float learning_rate,float adagrad_epsilon,float annuation_rate):
	BaseNetwork(input_size,output_size,batch_size,network_name,cublas,learning_rate,adagrad_epsilon,annuation_rate)
{
}

HiddenNetwork::~HiddenNetwork(){}

void HiddenNetwork::learningBackPropagation(mtk::MatrixXf &next_error, const mtk::MatrixXf &d2, const mtk::MatrixXf *w2){
	const float one = 1.0f,zero = 0.0f;
	mtk::MatrixFunction::map<dActReLU>(u1,u1);
	CUDA_HANDLE_ERROR(hipDeviceSynchronize());
	CUBLAS_HANDLE_ERROR(hipblasSgemm(cublas,HIPBLAS_OP_T,HIPBLAS_OP_N,
			u.getRows(),u.getCols(),d2.getRows(),
			&one,
			w2->getDevicePointer(),w2->getRows(),
			d2.getDevicePointer(),d2.getRows(),
			&zero,
			u.getDevicePointer(),u.getRows()));
	mtk::MatrixFunction::elementwiseProduct(cublas,next_error,u,u1);
	float alpha = 1.0f/batch_size;
	CUBLAS_HANDLE_ERROR(hipblasSgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_T,
			output_size,input_size,batch_size,
			&alpha,
			next_error.getDevicePointer(),next_error.getRows(),
			z0.getDevicePointer(),z0.getRows(),
			&zero,
			rdw1.getDevicePointer(),rdw1.getRows()));
	CUBLAS_HANDLE_ERROR(hipblasSgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N,
			output_size,1,batch_size,
			&alpha,
			next_error.getDevicePointer(),next_error.getRows(),
			all1_b.getDevicePointer(),batch_size,
			&zero,
			rdb1.getDevicePointer(),rdb1.getRows()));
}
void HiddenNetwork::learningActivation(mtk::MatrixXf &output, const mtk::MatrixXf &input) {
	mtk::MatrixFunction::map<ActReLU>(output,input);
}
void HiddenNetwork::testActivation(mtk::MatrixXf &output, const mtk::MatrixXf &input) {
	mtk::MatrixFunction::map<ActReLU>(output,input);
}

void HiddenNetwork::learningBackPropagation(mtk::MatrixXf &next_error, const mtk::MatrixXf &d2){
	BaseNetwork::learningBackPropagation(next_error,d2);
}
