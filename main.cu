#include <iostream>
#include "cuda_common.h"
#include "hiddenlayer.h"
#include "matrix_array.h"


int main(){
	hipblasHandle_t cublas;
	CUBLAS_HANDLE_ERROR(hipblasCreate(&cublas));
	mtk::HiddenLayer h0(100,100,100,"h",&cublas);
	//mtk::HiddenLayer h0(100,100,100,"h",&cublas,10.f,1.0f,1.0f);
	h0.learningReflect();
	CUBLAS_HANDLE_ERROR(hipblasDestroy( cublas));
}
