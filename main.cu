#include <iostream>
#include "cuda_common.h"
#include "hiddenlayer.h"
#include "matrix_array.h"
#include "softmaxlayer.h"

const int input_size = 28 * 28;
const int layer0_output_size = 20 * 20;
const int layer1_output_size = 10;
const int batch_size = 64;
const int calc = 200;
const int test_interval = 500;

int main(){
	hipblasHandle_t cublas;
	CUBLAS_HANDLE_ERROR(hipblasCreate(&cublas));

	// layers
	mtk::HiddenLayer layer0(input_size,layer0_output_size,batch_size,"layer0",cublas);
	mtk::SoftmaxLayer layer1(layer0_output_size,layer1_output_size,batch_size,"layer1",cublas);

	// feature
	mtk::MatrixXf input,hidden0,output;
	input.setSize(input_size,batch_size)->allocateDevice()->initDeviceRandom(-1.0f,1.0f);
	hidden0.setSize(layer0_output_size,batch_size)->allocateDevice()->initDeviceConstant(0.0f);
	output.setSize(layer1_output_size,batch_size)->allocateDevice()->allocateHost()->initDeviceConstant(0.0f);

	// error 
	mtk::MatrixXf input_error,hidden0_error,output_error;
	input_error.setSize(layer0_output_size,batch_size)->allocateDevice()->initDeviceConstant(0.0f);
	hidden0_error.setSize(layer1_output_size,batch_size)->allocateDevice()->initDeviceConstant(0.0f);
	output_error.setSize(layer1_output_size,batch_size)->allocateDevice()->initDeviceConstant(1.0f);
	for(int c = 0;c < calc;c++){
		// 順方向計算
		layer0.learningForwardPropagation(hidden0,input);
		layer1.learningForwardPropagation(output,hidden0);
		// 誤差計算

		// 逆方向計算
		layer1.learningBackPropagation(	hidden0_error, output_error);
		layer0.learningBackPropagation( input_error, hidden0_error, layer1.getWeightPointer());
		// 反映
		layer0.learningReflect();
		layer1.learningReflect();
		if((c+1)%test_interval == 0){std::cout<<c<<std::endl;}
	}
	CUBLAS_HANDLE_ERROR(hipblasDestroy( cublas));
}
