#include <iostream>
#include "cuda_common.h"
#include "hiddenlayer.h"
#include "matrix_array.h"


int main(){
	hipblasHandle_t cublas;
	CUBLAS_HANDLE_ERROR(hipblasCreate(&cublas));
	CUBLAS_HANDLE_ERROR(hipblasDestroy( cublas));
}
