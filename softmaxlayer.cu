#include "hip/hip_runtime.h"
#include "softmaxlayer.h"
#include "matrix_function.h"
#include "cuda_common.h"

using namespace mtk;

template<class T>
__global__ void deviceMap(float *device_ptr_dst,float* device_ptr_src,int max_t){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(max_t <= tid)
		return;
	device_ptr_dst[tid] = T()(device_ptr_src[tid]);
}

class Exp{
public:
	__device__ float operator()(float a) const{
		return expf(a);
	}
};
class Inverse{
public:
	__device__ float operator()(float a) const{
		return 1.0f/a;
	}
};

SoftmaxLayer::SoftmaxLayer(int input_size,int output_size,int batch_size,std::string layer_name,hipblasHandle_t cublas,float learning_rate,float adagrad_epsilon,float attenuation_rate):
	BaseLayer(input_size,output_size,batch_size,layer_name,cublas,learning_rate,adagrad_epsilon,attenuation_rate)
{
	input_row_0.setSize(1,batch_size)->allocateDevice()->initDeviceConstant(0.0f);
	inverse.setSize(output_size,batch_size)->allocateDevice()->initDeviceConstant(0.0f);
	output0.setSize(output_size,batch_size)->allocateDevice()->initDeviceConstant(0.0f);
}

SoftmaxLayer::~SoftmaxLayer(){}

void SoftmaxLayer::learningBackPropagation(mtk::MatrixXf& next_error,const mtk::MatrixXf& d2,const mtk::MatrixXf *w2){
	mtk::MatrixFunction::copy(cublas,next_error,d2);
	mtk::MatrixFunction::copy(cublas,d1,d2);
}

void SoftmaxLayer::activation(mtk::MatrixXf& output,const mtk::MatrixXf& input){
	//input行列の0行目を取り出す
	const float one = 1.0f,minus_one = -1.0f,zero = 0.0f;
	mtk::MatrixFunction::copy(cublas,output,input);
	/*CUBLAS_HANDLE_ERROR( hipblasScopy(cublas,output.getRows()*output.getCols(),
				input.getDevicePointer(),1,
				output.getDevicePointer(),1) );*/
	// 全列の要素からその列の先頭要素の値を引く
	CUBLAS_HANDLE_ERROR( hipblasScopy(cublas,batch_size,
				input.getDevicePointer(), output_size,
				input_row_0.getDevicePointer(),1));
	CUBLAS_HANDLE_ERROR( hipblasSgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N,
				output_size,batch_size,1,
				&minus_one,
				all1_o.getDevicePointer(),output_size,
				input_row_0.getDevicePointer(),1,
				&one,
				output.getDevicePointer(),output_size) );
	mtk::MatrixFunction::map<Exp>(output,output);
	// 和を取る
	CUBLAS_HANDLE_ERROR( hipblasSgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N,
				1,batch_size,output_size,
				&one,
				all1_o.getDevicePointer(),1,
				output.getDevicePointer(),output_size,
				&zero,
				input_row_0.getDevicePointer(),1));
	// 逆数を計算
	//deviceMap<Inverse><<<BLOCKS,threads_ceildiv(input_row_0.getSize(),BLOCKS)>>>(input_row_0.getDevicePointer(),input_row_0.getDevicePointer(),input_row_0.getSize());
	mtk::MatrixFunction::map<Inverse>(input_row_0,input_row_0);
	// 逆数の行列を計算
	CUBLAS_HANDLE_ERROR( hipblasSgemm( cublas, HIPBLAS_OP_N,HIPBLAS_OP_N,
				output_size,batch_size,1,
				&one,
				all1_o.getDevicePointer(),output_size,
				input_row_0.getDevicePointer(),1,
				&zero,
				inverse.getDevicePointer(),output_size) );
	mtk::MatrixFunction::elementwiseProduct(cublas,output0,output,inverse);
	/*CUBLAS_HANDLE_ERROR(hipblasSsbmv(cublas,HIPBLAS_FILL_MODE_LOWER,
			inverse.getCols()*inverse.getRows(),0,&one,
			inverse.getDevicePointer(),1,
			output.getDevicePointer(),1,
			&zero,output0.getDevicePointer(),1));*/
	mtk::MatrixFunction::copy(cublas,output,output0);
	/*CUBLAS_HANDLE_ERROR( hipblasScopy(cublas,output0.getRows()*output0.getCols(),
				output0.getDevicePointer(),1,
				output.getDevicePointer(),1) );*/
	//output.copyToHost();
	//output.print("output");
	//input.print("input");
}
