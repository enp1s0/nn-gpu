#include "cuda_event.h"
#include "cuda_common.h"

using namespace mtk;

CudaEvent* CudaEvent::createEvent(std::string event_name){
	hipEvent_t event;
	CUDA_HANDLE_ERROR( hipEventCreate( &event ) );
	events_map.insert(std::make_pair(event_name,event));
	return this;
}

float CudaEvent::elapsedTime(std::string start_event,std::string stop_event){
	float elapsed_time;
	CUDA_HANDLE_ERROR( hipEventElapsedTime( &elapsed_time, events_map[start_event], events_map[stop_event] ) );
	return elapsed_time;
}

void CudaEvent::recordEvent(std::string event_name){
	CUDA_HANDLE_ERROR( hipEventRecord( events_map[event_name], 0) );
	CUDA_HANDLE_ERROR( hipEventSynchronize( events_map[event_name] ) );
}
