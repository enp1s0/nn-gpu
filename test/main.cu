#include "hip/hip_runtime.h"
#include "matrix_array.h"
#include "cublas_common.h"
#include "matrix_functions.h"
#include <iostream>
const int BLOCKS = 1 << 7;

void showMatrix(mtk::MatrixXf &mat0){
	for(int i = 0;i < mat0.getRows();i++){
		for(int j = 0;j < mat0.getCols();j++){
			std::cout<<mat0.getHostPointer()[i+j*mat0.getRows()]<<" ";
		}
		std::cout<<std::endl;
	}
}
template<class T>
__global__ void deviceMap(float *device_ptr_dst,float* device_ptr_src,int max_t){
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(max_t <= tid)
		return;
	device_ptr_dst[tid] = T()(device_ptr_src[tid]);
}
class POI{
public:
	__device__ float operator()(float a){
		return a*a;
	}
};

void unary(){
	mtk::MatrixXf mat0,mat1;
	mat0.setSize(10,5)->allocateDevice()->allocateHost()->initDeviceRandom(-1.0f,1.0f);
	mat1.setSize(10,5)->allocateDevice()->allocateHost()->initDeviceConstant(0.0f);

	deviceMap<POI><<<BLOCKS,(mat0.getCols()*mat0.getRows()+BLOCKS-1)/BLOCKS>>>(mat1.getDevicePointer(),mat0.getDevicePointer(),mat0.getCols() * mat0.getRows());

	mat0.copyToHost();
	mat1.copyToHost();

	std::cout<<"mat0"<<std::endl;
	showMatrix(mat0);
	std::cout<<"mat1"<<std::endl;
	showMatrix(mat1);
}
void element_wise(hipblasHandle_t cublas){
	mtk::MatrixXf mat0,mat1,mat2;
	mat0.setSize(10,5)->allocateDevice()->allocateHost()->initDeviceRandom(-1.0f,1.0f);
	mat1.setSize(10,5)->allocateDevice()->allocateHost()->initDeviceRandom(-1.0f,1.0f);
	mat2.setSize(10,5)->allocateDevice()->allocateHost()->initDeviceRandom(-1.0f,1.0f);
	/*const float one = 1.0f,zero = 0.0f;
	CUBLAS_HANDLE_ERROR( hipblasSsbmv( cublas, HIPBLAS_FILL_MODE_LOWER,
				mat0.getRows()*mat0.getCols(),0,&one,
				mat0.getDevicePointer(),1,
				mat1.getDevicePointer(),1,
				&zero,mat2.getDevicePointer(),1));*/
	mtk::CublasFunction::elementwiseProduct(cublas,mat2,mat0,mat1);
	mat0.copyToHost();
	mat1.copyToHost();
	mat2.copyToHost();
	std::cout<<"mat0"<<std::endl;
	showMatrix(mat0);
	std::cout<<"mat1"<<std::endl;
	showMatrix(mat1);
	std::cout<<"mat2"<<std::endl;
	showMatrix(mat2);
}

int main(){
	hipblasHandle_t cublas;
	CUBLAS_HANDLE_ERROR(hipblasCreate(&cublas));
	element_wise(cublas);
	//unary();
	CUBLAS_HANDLE_ERROR(hipblasDestroy( cublas));
}

